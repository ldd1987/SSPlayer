#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "common/inc/hip/hip_vector_types.h"
#include "cudahdr.h"
#include <stdio.h>
#include <vector>
const int blockperthread = 512;
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x +  blockIdx.x * blockperthread;
    c[i] = a[i] + b[i];
}



__host__ __device__
float4 mul(const float4 *m, const float4 &v)
{
	float4 r;
	r.x = dot(v, m[0]);
	r.y = dot(v, m[1]);
	r.z = dot(v, m[2]);
	r.w = 1.0f;
	return r;
}

__host__ __device__
float4 mul(const float4 &v, const float4 *m)
{
	float4 r;
	r.x = dot(v, m[0]);
	r.y = dot(v, m[1]);
	r.z = dot(v, m[2]);
	r.w = 1.0f;
	return r;
}

__host__ __device__  float4 hable(float4 x)
{
	const float A = 0.15, B = 0.50, C = 0.10, D = 0.20, E = 0.02, F = 0.30;
	return ((x * (A*x + (C*B)) + (D*E)) / (x * (A*x + B) + (D*F))) - E / F;
}

__host__ __device__ float4 HDRToneMapping(float4 rgb, float LuminanceScale)
{
	float4 HABLE_DIV = hable(make_float4(11.2,11.2,11.2,11.2));
	float4 rgba = hable(rgb* LuminanceScale) / HABLE_DIV;
	return rgba;
}

__host__ __device__ float4 transformPrimaries(float4 rgb, int primaries,int disprimaries, float4*TransPrimaries)
{
	if (primaries != disprimaries)
	{
		return fmaxf(mul(rgb, TransPrimaries),make_float4(0,0,0 ,0));
	}
	else
	{
		return rgb;
	}
}

__host__ __device__ float4 toneMapping(float4 rgb, int  transfer,int distransfer, float LuminanceScale)
{
	if (distransfer == transfer)
	{
		return rgb;
	}
	if (distransfer == 1 || distransfer == 4)
	{
		if (transfer == 16 || transfer == 18)
		{
			return HDRToneMapping(rgb, LuminanceScale);
		}
		else
		{
			return rgb * LuminanceScale;
		}
	}
	else
	{
		return rgb * LuminanceScale;
	}

}



// 电光转换函数  这里转换为自然光 
__host__ __device__ float inverse_HLG(float x)
{
	const float B67_a = 0.17883277;
	const float B67_b = 0.28466892;
	const float B67_c = 0.55991073;
	const float B67_inv_r2 = 4.0;
	if (x <= 0.5)
		x = x * x * B67_inv_r2;
	else
		x = exp((x - B67_c) / B67_a) + B67_b;
	return x;
}

// 自然光转换为电信号
__host__ __device__ float LineToHLG(float Lc)
{
	const double a = 0.17883277;
	const double b = 0.28466892;
	const double c = 0.55991073;
	return (0.0 > Lc) ? 0.0 :
		(Lc <= 1.0 / 12.0 ? sqrt(3.0 * Lc) : a * log(12.0 * Lc - b) + c);
}


__host__ __device__  float4 ST2084TOLinear(float4 rgb)
{
	const float ST2084_m1 = 2610.0 / (4096.0 * 4);
	const float ST2084_m2 = (2523.0 / 4096.0) * 128.0;
	const float ST2084_c1 = 3424.0 / 4096.0;
	const float ST2084_c2 = (2413.0 / 4096.0) * 32.0;
	const float ST2084_c3 = (2392.0 / 4096.0) * 32.0;
	rgb = fpowf(fmaxf(rgb, make_float4(0,0,0, 0)), 1.0 / ST2084_m2);
	rgb = fmaxf(rgb - ST2084_c1, make_float4(0,0,0,0)) / (ST2084_c2 - ST2084_c3 * rgb);
	rgb = fpowf(rgb, 1.0 / ST2084_m1);
	return rgb * 10000;
}


// 这里转换为自然 光 
__host__ __device__  float4 HLGTOSenceLinear(float4 rgb)
{
	rgb.x= inverse_HLG(rgb.x);
	rgb.y = inverse_HLG(rgb.y);
	rgb.z = inverse_HLG(rgb.z);
	return rgb;
}

// 这里转换为display 光 
__host__ __device__  float4 HLGTOLinear(float4 rgb)
{
	const float alpha_gain = 2000;
	rgb.x = inverse_HLG(rgb.x);
	rgb.y = inverse_HLG(rgb.y);
	rgb.z = inverse_HLG(rgb.z);
	// 下面是光光转换函数 转换为display 光
	float3 ootf_2020 = make_float3(0.2627, 0.6780, 0.0593);
	float ootf_ys = alpha_gain * dot(ootf_2020, make_float3(rgb.x, rgb.y, rgb.z));
	return rgb * powf(ootf_ys, 1.200 - 1.0);
}
__host__ __device__  float4 BT709TOLinear(float4 rgb)
{
	return fpowf(rgb, 1.0 / 0.45);
}

__host__ __device__  float4 BT470M_SRGB_TOLinear(float4 rgb)
{
	return fpowf(rgb, 2.2);
}
__host__ __device__  float4 BT470BGTOLinear(float4 rgb)
{
	return fpowf(rgb, 2.8);
}

__host__ __device__  float4 LineTOSRGB(float4 rgb)
{
	return fpowf(rgb, 1.0 / 2.2);
}
__host__ __device__  float4 LineTOST2084(float4 rgb)
{
	const float ST2084_m1 = 2610.0 / (4096.0 * 4);
	const float ST2084_m2 = (2523.0 / 4096.0) * 128.0;
	const float ST2084_c1 = 3424.0 / 4096.0;
	const float ST2084_c2 = (2413.0 / 4096.0) * 32.0;
	const float ST2084_c3 = (2392.0 / 4096.0) * 32.0;
	rgb = fpowf(rgb / 10000, ST2084_m1);
	rgb = (ST2084_c1 + ST2084_c2 * rgb) / (1 + ST2084_c3 * rgb);
	rgb = fpowf(rgb, ST2084_m2);
	return rgb;
}


__host__ __device__  float4 PQToHLG(float4 rgb, float LuminanceScale)
{

	float r = rgb.x / LuminanceScale / 10000.0 * 10;
	float g = rgb.y / LuminanceScale / 10000.0 * 10;
	float b = rgb.z / LuminanceScale / 10000.0 * 10;
	 r = r > 1.0 ? 1.0 : r;
	 g = g > 1.0 ? 1.0 : g;
	 b = b > 1.0 ? 1.0 : b;
	float a = rgb.w;
	// 上面得到的是display光 
	// 逆光光转换函数  将display 光转换为自然光 
	if (0)
	{
		float3 ootf_2020 = make_float3(0.2627, 0.6780, 0.0593);
		float ootf_ys = dot(ootf_2020, make_float3(rgb.x, rgb.y,rgb.z));
		ootf_ys = powf(ootf_ys, (1.0 - 1.2) / 1.200);
		float hlgr = r * ootf_ys;
		float hlgg = g * ootf_ys;
		float hlgb = b * ootf_ys;
		float hlga = a;
		hlgr = LineToHLG(hlgr);
		hlgg = LineToHLG(hlgg);
		hlgb = LineToHLG(hlgb);
		return  make_float4(hlgr, hlgg, hlgb, hlga);
	}
	else
	{
		r = LineToHLG(r);
		g = LineToHLG(g);
		b = LineToHLG(b);
		return  make_float4(r, g, b, a);
	}


}

__host__ __device__  float4 SDRToHLG(float4 rgb, float LuminanceScale)
{
	float a = rgb.w;
	rgb.x = 0.265 *rgb.x * 2/ LuminanceScale;
	rgb.y = 0.265 *rgb.y * 2 / LuminanceScale;
	rgb.z = 0.265 *rgb.z * 2 / LuminanceScale;
	float ootf_ys = 1;
	float r = rgb.x > 1.0 ? 1.0 : rgb.x;
	float g = rgb.y > 1.0 ? 1.0 : rgb.y;
	float b = rgb.z > 1.0 ? 1.0 : rgb.z;
	float hlgr = r * ootf_ys;
	float hlgg = g * ootf_ys;
	float hlgb = b * ootf_ys;
	hlgr = LineToHLG(hlgr);
	hlgg = LineToHLG(hlgg);
	hlgb = LineToHLG(hlgb);
	float hlga = a;
	return  make_float4(hlgr, hlgg, hlgb, hlga);
}

__host__ __device__  float4 LinerToSTDB67(float4 hlg, int transfer, int distransfer, float LuminanceScale)
{
	if (transfer == distransfer)
	{
		return hlg;
	}
	else
	{
		if (transfer == 16)
		{
			return PQToHLG(hlg, LuminanceScale);
		}
		else
		{
			return SDRToHLG(hlg, LuminanceScale);
		}
	}


}

__host__ __device__  float4 linearToDisplay(float4 rgb, int transfer, int distransfer, float LuminanceScale)
{
	if (distransfer == transfer)
	{
		return  rgb;
	}
	else if (distransfer == 16)
	{
		return LineTOST2084(rgb);
	}
	else if (distransfer == 18)
	{
		return LinerToSTDB67(rgb, transfer, distransfer, LuminanceScale);
	}
	else if (distransfer == 1)
	{
		return fpowf(rgb,1.0/2.2);
	}
	else if (distransfer == 4)
	{
		return fpowf(rgb, 1.0 / 2.2);
	}
	else
	{
		return rgb;
	}
}

__host__ __device__  float4 sourcetolinekernel(float4 rgb,  int transfer, int distransfer)
{
	if (transfer == distransfer)
	{
		return rgb;
	}
	else if (transfer == 8)  //line 
	{
		return rgb;
	}
	else if (transfer == 16) // pq
	{
		return ST2084TOLinear(rgb);
	}
	else if (transfer == 18) // hlg
	{
		return HLGTOLinear(rgb);
	}
	else if (transfer == 1) // bt709
	{
		return BT709TOLinear(rgb);
	}
	else if (transfer == 4)
	{
		return BT470M_SRGB_TOLinear(rgb);
	}
	else if (transfer == 5)
	{
		return BT470BGTOLinear(rgb);
	}
	else
	{
		return rgb;
	}
}

__host__ __device__  float4 Render2RGBA(float4 rgb, int transfer, int distransfer, int primary, int disprimary, float LuminanceScale, float4 *TransPrimaries)
{
	float a = rgb.w;
	rgb.w = 0;
	rgb = sourcetolinekernel(rgb, transfer, distransfer);
	rgb = transformPrimaries(rgb, primary, disprimary, TransPrimaries);
	rgb = toneMapping(rgb, transfer, distransfer, LuminanceScale);
	rgb = linearToDisplay(rgb, transfer, distransfer, LuminanceScale);
	rgb.w = a;
	return rgb;
}
__host__ __device__  float4 RGBA2yuv(float4 rgb, float4 *rgb2yuv)
{
	float4 yuva = mul(rgb, rgb2yuv);
	float y = clamp(yuva.x, 0.0, 1.0);
	float u = clamp(yuva.y, 0.0, 1.0);
	float v = clamp(yuva.z, 0.0, 1.0);
	return make_float4(y, u, v, 1.0);
}

__host__ __device__  float4 yuv2rgb(float4 yuv, float4 *yuv2rgb)
{
	float4 yuva = mul(yuv, yuv2rgb);
	float r = clamp(yuva.x, 0.0, 1.0);
	float g = clamp(yuva.y, 0.0, 1.0);
	float b = clamp(yuva.z, 0.0, 1.0);
	return make_float4(r, g, b, 1.0);
}

__global__ void yuv420p10torgbakernel(unsigned short *dst, const unsigned short *y, const unsigned short *u, const unsigned short *v, const float4 *matrix,const float4 *white, int width,int height)
{
	int index = threadIdx.x + blockIdx.x * blockperthread;
	int xpos = index % width;
	int ypos = index / width ;
	xpos = xpos / 2;
	ypos = ypos / 2;
	int uvindex = xpos + ypos * width / 2;
	float y0 = y[index]/1023.0;
	float u0 = u[uvindex] / 1023.0;
	float v0 = v[uvindex] / 1023.0;
	 float4 rgba = mul(make_float4(y0,u0,v0,1), white);
	 rgba = fmaxf(mul(rgba, matrix), make_float4(0,0,0,0));
	 float r0 = clamp(rgba.x, 0.0, 1.0);;
	 float g0 = clamp(rgba.y, 0.0, 1.0);;
	 float b0 = clamp(rgba.z, 0.0, 1.0);
	 dst[index * 4 + 0] = (unsigned short)(r0 *1023);
	 dst[index * 4 + 1] = (unsigned short)(g0 * 1023);
	 dst[index * 4 + 2] = (unsigned short)(b0 * 1023);
	 dst[index * 4 + 3] = 1 * 1023;

}

__global__ void yuvnv12p10torgbakernel(unsigned short *rgba, const unsigned short *y, const unsigned short *u, const unsigned short *v, const float *matrix, int width, int height)
{
	int index = threadIdx.x + blockIdx.x * blockperthread;
	int xpos = index % width;
	int ypos = index / (width *height);
	xpos = xpos / 2;
	ypos = ypos / 2;
	int uvindex = xpos + ypos * height / 2;
	float y0 = y[index] / 1023.0;
	float u0 = u[uvindex] / 1023.0;
	float v0 = v[uvindex] / 1023.0;
	float r0 = matrix[0] * y0 + matrix[1] * u0 + matrix[2] * v0 + matrix[3];
	float g0 = matrix[4] * y0 + matrix[5] * u0 + matrix[6] * v0 + matrix[7];
	float b0 = matrix[8] * y0 + matrix[9] * u0 + matrix[10] * v0 + matrix[11];
	r0 = clamp(r0, 0.0, 1.0);
	g0 = clamp(g0, 0.0, 1.0);
	b0 = clamp(b0, 0.0, 1.0);
	rgba[index * 4 + 0] = r0 * 1023;
	rgba[index * 4 + 1] = g0 * 1023;
	rgba[index * 4 + 2] = v0 * 1023;

}

__global__ void yuv444p10torgbkernel(unsigned short *rgba, const unsigned short *y, const unsigned short *u, const unsigned short *v, const float *matrix, int width, int height)
{
	int index = threadIdx.x + blockIdx.x * blockperthread;
	int uvindex = index;
	float y0 = y[index] / 1023.0;
	float u0 = u[uvindex] / 1023.0;
	float v0 = v[uvindex] / 1023.0;
	float r0 = matrix[0] * y0 + matrix[1] * u0 + matrix[2] * v0 + matrix[3];
	float g0 = matrix[4] * y0 + matrix[5] * u0 + matrix[6] * v0 + matrix[7];
	float b0 = matrix[8] * y0 + matrix[9] * u0 + matrix[10] * v0 + matrix[11];
	r0 = clamp(r0, 0.0, 1.0);
	g0 = clamp(g0, 0.0, 1.0);
	b0 = clamp(b0, 0.0, 1.0);
	rgba[index * 4 + 0] = r0 * 1023;
	rgba[index * 4 + 1] = g0 * 1023;
	rgba[index * 4 + 2] = v0 * 1023;
}
__global__ void yuv422p10torgbakernel(unsigned short *rgba, const unsigned short *y, const unsigned short *u, const unsigned short *v, const float *matrix, int width, int height)
{
	int index = threadIdx.x + blockIdx.x * blockperthread;
	int xpos = index % width;
	int ypos = index / width;
	xpos = xpos / 2;
	int uvindex = xpos + ypos * height;
	float y0 = y[index] / 1023.0;
	float u0 = u[uvindex] / 1023.0;
	float v0 = v[uvindex] / 1023.0;
	float r0 = matrix[0] * y0 + matrix[1] * u0 + matrix[2] * v0 + matrix[3];
	float g0 = matrix[4] * y0 + matrix[5] * u0 + matrix[6] * v0 + matrix[7];
	float b0 = matrix[8] * y0 + matrix[9] * u0 + matrix[10] * v0 + matrix[11];
	r0 = clamp(r0, 0.0, 1.0);
	g0 = clamp(g0, 0.0, 1.0);
	b0 = clamp(b0, 0.0, 1.0);
	rgba[index * 4 + 0] = r0 * 1023;
	rgba[index * 4 + 1] = g0 * 1023;
	rgba[index * 4 + 2] = v0 * 1023;
}


__global__ void rgbtoyuv420p10(int *c, const int *a, const float *b)
{
	int i = threadIdx.x + blockIdx.x * blockperthread;
	c[i] = a[i] + b[i];
}
__global__ void rgbtoyuv444p10(int *c, const int *a, const float *b)
{
	int i = threadIdx.x + blockIdx.x * blockperthread;
	c[i] = a[i] + b[i];
}

__global__ void rgba10resize(unsigned short *dst, const unsigned short *src, int srcwidth,int srcheight,int dstwidth,int dstheight,int xstart,int ystart, float scale)
{
	int i = threadIdx.x + blockIdx.x * blockperthread;
	int xpos = i % dstwidth;
	int ypos = i / dstwidth;

	float xposorg = xpos * scale;
	float yposorg = ypos * scale;
	int xx = xposorg;
	int yy = yposorg;
	float r = src[yy * srcwidth * 4 + xx *4 + 0]/1023;
	float g = src[yy * srcwidth * 4 + xx * 4 + 1] / 1023;
	float b = src[yy * srcwidth * 4 + xx * 4 + 2] / 1023;
	float a = src[yy * srcwidth * 4 + xx * 4 + 3] / 1023;
	dst[ystart * dstwidth * 4 + xstart *4 + i * 4 + 0] = r * 1023;
	dst[ystart * dstwidth * 4 + xstart * 4 + i * 4 + 1] = g*1023;
	dst[ystart * dstwidth * 4 + xstart * 4 + i * 4 + 2] = b*1023;
	dst[ystart * dstwidth * 4 + xstart * 4 + i * 4 + 3] = a*1023;
}

__global__ void rgba10resize2YUV42010(unsigned short *dsty, unsigned short *dstuv, const unsigned short *src, int srcwidth, int srcheight, 
	int dstwidth, int dstheight, int xstart, int ystart, float scale, int transfer, int distransfer, int primary, int disprimary,
	float LuminanceScale, float4 *TransPrimaries,float4 *rgb2yuv)
{
	int i = threadIdx.x + blockIdx.x * blockperthread;
	if (i >= dstwidth * dstheight)
	{
		return;
	}
	int xpos = i % dstwidth;
	int ypos = i / dstwidth;
	float xposorg = xpos * scale;
	float yposorg = ypos * scale;
	int xx = xposorg;
	int yy = yposorg;
	if (xx >= srcwidth || yy >= srcheight)
	{
		return;
	}
	float r = src[yy * srcwidth * 4 + xx * 4 + 0] / 1023.0;
	float g = src[yy * srcwidth * 4 + xx * 4 + 1] / 1023.0;
	float b = src[yy * srcwidth * 4 + xx * 4 + 2] / 1023.0;
	float4 rgba = make_float4(r, g, b, 1);
	rgba = Render2RGBA(rgba, transfer, distransfer, primary, disprimary, LuminanceScale, TransPrimaries);
	float4 yuva = RGBA2yuv(rgba, rgb2yuv);
	dsty[i] = clamp(yuva.x, 0.0, 1.0) * 1023;
	if (1)
	{
		//float a = src[yy * srcwidth * 4 + xx * 4 + 3] / 1023.0;
		
		if (xpos % 2 == 0 && ypos % 2 == 0)
		{
			dstuv[ypos * dstwidth / 2 + xpos + 0] = clamp(yuva.y, 0.0, 1.0) * 1023;
			dstuv[ypos * dstwidth / 2 + xpos + 1] = clamp(yuva.z, 0.0, 1.0) * 1023;
		}
	}
	
}


void yuv420p10tonv12p10resize(unsigned short *NV12, const unsigned short *y,
	const unsigned short *u, const unsigned short *v, const float *yuv2rgb, const float *white, float * TransferPirmary, const float *rgb2yuv, int srcwidth, int srcheight,
	int dstwidth, int dstheight, int transfer, int distranfer, int primary, int disprimary, float luama)
{
	int nGpu = 0;
	hipError_t res = hipGetDeviceCount(&nGpu);
	if (res != hipSuccess)
	{
		fprintf(stderr, "addWithCuda failed!");
		return;
	}
	std::vector< hipDeviceProp_t> vProp;
	for (int i = 0; i < nGpu; i++)
	{
		hipDeviceProp_t deviceprop;
		res = hipGetDeviceProperties(&deviceprop, i);
		if (res == hipSuccess)
		{
			vProp.push_back(deviceprop);
		}
	}
	hipError_t cudaStatus;
	unsigned short * ydevice = 0;
	unsigned short *udevice = 0;
	unsigned short *vdevice = 0;
	unsigned short *rgbdevice = 0;
	unsigned short *nv1210device = 0;
	float4 *yuv2rgbmatrixdevice = 0;
	float4 *whitedevice = 0;
	float4 *rgb2yuvmatrixdevice = 0;
	float4 *transferprimarymatrixdevice = 0;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&yuv2rgbmatrixdevice, 4 * sizeof(float4));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&transferprimarymatrixdevice, 4 * sizeof(float4));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&rgb2yuvmatrixdevice, 4 * sizeof(float4));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&whitedevice, 4 * sizeof(float4));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&rgbdevice, srcwidth *srcheight * 4 * sizeof(unsigned short));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&nv1210device, dstwidth *dstheight * 3/2 * sizeof(unsigned short));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	


	cudaStatus = hipMalloc((void**)&ydevice, srcwidth *srcheight * sizeof(unsigned short));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&udevice, srcwidth *srcheight / 4 * sizeof(unsigned short));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&vdevice, srcwidth *srcheight / 4 * sizeof(unsigned short));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(yuv2rgbmatrixdevice, yuv2rgb, 16 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(rgb2yuvmatrixdevice, rgb2yuv, 16 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(transferprimarymatrixdevice, TransferPirmary, 16 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(whitedevice, white, 16 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(ydevice, y, srcwidth *srcheight * sizeof(unsigned short), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(udevice, u, srcwidth *srcheight / 4 * sizeof(unsigned short), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(vdevice, v, srcwidth *srcheight / 4 * sizeof(unsigned short), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	int nsize = srcwidth * srcheight;
	unsigned int grid = (nsize + blockperthread - 1) / blockperthread;
	dim3 ngrid = { grid };
	unsigned int perthread = blockperthread;
	dim3 nthread = { perthread };
	// Launch a kernel on the GPU with one thread for each element.
	yuv420p10torgbakernel << <grid, perthread >> > (rgbdevice, ydevice, udevice, vdevice, yuv2rgbmatrixdevice, whitedevice, srcwidth, srcheight);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	 nsize = dstwidth * dstheight;
	 grid = (nsize + blockperthread - 1) / blockperthread;
	 ngrid = { grid };
	 unsigned short *dsty = nv1210device;
	 unsigned short *dstuv = nv1210device + dstwidth * dstheight;
	 float scale = 1.0 * srcwidth / dstwidth;
	rgba10resize2YUV42010 << <grid, perthread >> > (dsty, dstuv,rgbdevice, srcwidth,srcheight,dstwidth,dstheight,0,0,scale,transfer,distranfer,
		primary, disprimary,luama,transferprimarymatrixdevice, rgb2yuvmatrixdevice);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(NV12, nv1210device, dstheight *dstwidth * 3 / 2* sizeof(unsigned short), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(rgbdevice);
	hipFree(nv1210device);
	hipFree(ydevice);
	hipFree(udevice);
	hipFree(vdevice);
	hipFree(yuv2rgbmatrixdevice);
	hipFree(rgb2yuvmatrixdevice);
	hipFree(whitedevice);
	hipFree(transferprimarymatrixdevice);
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return;
	}
}
void yuv420p10torgba(unsigned short *rgba, const unsigned short *y,
	const unsigned short *u, const unsigned short *v, const float *matrix, const float *white,int width, int height)
{
	int nGpu = 0;
	hipError_t res = hipGetDeviceCount(&nGpu);
	if (res != hipSuccess)
	{
		fprintf(stderr, "addWithCuda failed!");
		return ;
	}
	std::vector< hipDeviceProp_t> vProp;
	for (int i = 0; i < nGpu; i++)
	{
		hipDeviceProp_t deviceprop;
		res = hipGetDeviceProperties(&deviceprop, i);
		if (res == hipSuccess)
		{
			vProp.push_back(deviceprop);
		}
	}
	hipError_t cudaStatus;
	unsigned short * ydevice = 0;
	unsigned short *udevice = 0;
	unsigned short *vdevice = 0;
	unsigned short *rgbdevice = 0;
	float4 *matrixdevice = 0;
	float4 *whitedevice = 0;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&matrixdevice,4 * sizeof(float4));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&whitedevice, 4 * sizeof(float4));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&rgbdevice, width *height *4* sizeof(unsigned short));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&ydevice, width *height * sizeof(unsigned short));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&udevice, width *height/4 * sizeof(unsigned short));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&vdevice, width *height / 4 * sizeof(unsigned short));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(matrixdevice, matrix, 16 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(whitedevice, white, 16 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(ydevice, y, width *height * sizeof(unsigned short), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(udevice, u, width *height / 4 * sizeof(unsigned short), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(vdevice, v, width *height / 4 * sizeof(unsigned short), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	int nsize = width * height;
	unsigned int grid = (nsize + blockperthread - 1) / blockperthread;
	dim3 ngrid = { grid};
	unsigned int perthread = blockperthread;
	dim3 nthread = { perthread };
	// Launch a kernel on the GPU with one thread for each element.
	yuv420p10torgbakernel << <grid, perthread >> > (rgbdevice,ydevice,udevice,vdevice, matrixdevice,whitedevice, width, height);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(rgba, rgbdevice, width *height * 4 * sizeof(unsigned short), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(rgbdevice);
	hipFree(ydevice);
	hipFree(udevice);
	hipFree(vdevice);
	hipFree(matrixdevice);
	hipFree(whitedevice);
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return ;
	}

}
#ifdef EXE
const long arraySize = 3840 * 2160;
int main()
{
	int nGpu = 0;
	hipError_t res = hipGetDeviceCount(&nGpu);
	if (res != hipSuccess)
	{
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	std::vector< hipDeviceProp_t> vProp;
	for (int i = 0 ;i < nGpu; i++)
	{
		hipDeviceProp_t deviceprop;
		res = hipGetDeviceProperties(&deviceprop, i);
		if (res == hipSuccess)
		{
			vProp.push_back(deviceprop);
		}
	}
	
     int *a = new int[arraySize];
	 int *b = new int[arraySize];
	 int *c = new int[arraySize];
	 for (int i = 0; i < arraySize; i++)
	 {
		 a[i] = 0;
		 b[i] = 1;
		 c[i] = 0;
	 }
  

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
	for (int i = 0; i < arraySize; i++)
	{
		if (c[i] != a[i] + b[i])
		{
			fprintf(stderr, "addWithCuda error----id is:%d!", i);
		}
	}
   

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	dim3 ngrid = { (arraySize + blockperthread-1) / blockperthread , 1,1};
	int perthread = blockperthread;
	dim3 nthread = { 512,1,1 };
    // Launch a kernel on the GPU with one thread for each element.
    addKernel<< <ngrid, nthread >> >(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
#endif